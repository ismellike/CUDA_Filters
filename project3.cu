#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <stdio.h>
#include "hip/hip_runtime.h"

using namespace std;

#include "ImageReader.h"
#include "ImageWriter.h"

__global__ void ApplyFilter(unsigned char* data, int width, int height, int channels, float* filter, int filterSize, unsigned char* outdata)
{
	int row = threadIdx.x;
	int mid = filterSize / 2;

	//establish shared filter
	__shared__ float sharedFilter[256];

	if(row < filterSize * filterSize)
		sharedFilter[row] = filter[row];
	
	__syncthreads();

	//imagereader gives RGBA
	for (int i = 0; i < width; i++) 
	{
		//only interested in RGB
		float accumulator[4] = { 0, 0, 0, 255 };

		//filter must be odd find mid
		for(int y = -mid; y <= mid; y++)
			for (int x = -mid; x <= mid; x++) 
			{
				//check targetLoc is valid
				if (row + y < 0 || row + y >= height || i + x < 0 || i + x >= width)
				{
					//out of bounds use other values
					//CASE 1: missing pixels with value 0 so skip
					continue;
				}
				//convolution
				int targetLoc = (row - y) * width * channels + (i - x) * channels;
				int filterLoc = (y + mid) * filterSize + (x + mid);
				
				for (int channel = 0; channel < 3; channel++)
					accumulator[channel] += data[targetLoc + channel] * sharedFilter[filterLoc];
			}

		//check valid values
		for (int channel = 0; channel < channels; channel++) 
		{
			if (accumulator[channel] < 0)
				accumulator[channel] = 0;
			else if (accumulator[channel] > 255)
				accumulator[channel] = 255;

			//set outdata
			outdata[row * width * channels + i * channels + channel] = accumulator[channel];
		}
	}
}

//create array of size N^2 and return N
int ReadFilter(string filterPath, float* packedArray) 
{
	ifstream inFile(filterPath);

	if (!inFile.is_open())
	{
		cout << "Invalid Filter" << endl;
		return 0;
	}

	if (packedArray)
		delete packedArray;

	int N = 0;
	int i = 0;
	int sum = 0;

	inFile >> N;
	packedArray = new float[N * N];
	while (i < N * N) 
	{
		inFile >> packedArray[i];
		sum += packedArray[i];
		i++;
	}

	if (sum > 1) 
	{
		for(i = 0; i < N*N; i++)
			packedArray[i] /= sum;

		return N;
	}

	if (sum < 0 || (sum != 0 && sum != 1))
	{
		cout << "Invalid filter" << endl;
		return 0;
	}

	return N;
}

int main(int argc, char* argv[])
{
	if (argc < 4) {
		cout << "USAGE: project3 [picture] [filter] [output]" << endl;
		return -1;
	}
	string filePath = argv[1], filterPath = argv[2], outputPath = argv[3];

	//get filter
	float* filter = nullptr;
	int N = ReadFilter(filterPath, filter);

	//establish reader
	ImageReader* reader = ImageReader::create(filePath);
	if (reader == nullptr)
	{
		cout << "Invalid reader" << endl;
		return -1;
	}

	//read basic info
	int xRes = reader->getWidth();
	int yRes = reader->getHeight();
	int nChannels = reader->getNumChannels();

	//get picture data
	cryph::Packed3DArray<unsigned char>* packedArray = reader->getInternalPacked3DArrayImage();
	unsigned char* h_data = packedArray->getModifiableData();
	int data_count = packedArray->getTotalNumberElements();

	//send data to device
	size_t data_size = sizeof(unsigned char) * data_count;
	unsigned char* d_data;
	unsigned char* d_outdata;

	hipMalloc((void**)&d_data, data_size);
	hipMalloc((void**)&d_outdata, data_size);
	hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice);

	//send filter to device
	size_t filter_size = sizeof(float) * N * N;
	float* d_filter;

	hipMalloc((void**)&d_filter, filter_size);
	hipMemcpy(d_filter, filter, filter_size, hipMemcpyHostToDevice);
	//launch kernel
	ApplyFilter<<< 1, yRes >>>(d_data, xRes, yRes, nChannels, d_filter, N, d_outdata);
	hipDeviceSynchronize();

	//read kernel data
	unsigned char* h_outdata = new unsigned char[data_count];
	hipMemcpy(h_outdata, d_outdata, data_size, hipMemcpyDeviceToHost);

	//establish writer
	ImageWriter* writer = ImageWriter::create(outputPath, xRes, yRes, nChannels);
	if (writer == nullptr)
	{
		cout << "Invalid writer" << endl;
		return -1;
	}

	//write image
	writer->writeImage(h_outdata);
	writer->closeImageFile();

	//free memory
	delete reader, writer;
	hipFree(d_data);
	hipFree(d_filter);
	hipFree(d_outdata);
	delete filter;
	delete h_outdata;
	return 0;
}